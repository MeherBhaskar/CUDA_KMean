
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


//TODO : used sharedvar to have (xi - yi)^2 generated in ||lel

__global__ void updateMeans(float *means, float *entries, int *closestMean, int num_entries, int num_means, int num_attribs)
{
	int id = threadIdx.y;
	int thisMeanCount = 0;
	float temp[100];
	for(int j = 0; j < num_attribs; j++)
	{
		temp[j] = 0;
	}
	for(int i = 0; i < num_entries; i++)
	{
		if(closestMean[i] == id)
		{
			//printf("Entry : %d :: Closest : %d \n", i, id);

			for(int j = 0; j < num_attribs; j++)
			{
				//printf("Adding %f to mean %d, attrib %d \n",entries[i*num_attribs + j],closestMean[i],j);
				temp[j]+=entries[i*num_attribs + j];
			}
			thisMeanCount++;
		}
	}
	
	for(int i = 0 ; i < num_attribs; i++)
	{
		if(thisMeanCount != 0)
		{
			
			means[id*num_attribs + i] = temp[i]/thisMeanCount;
			printf("mean : %d , attrib : %d , newMean : %f, count : %d \n",id, i,means[id*num_attribs + i],thisMeanCount);
		}
	}
}

__global__ void getClosestMeans(float *means, float *entries, int *closestMean, int num_entries, int num_means, int num_attribs, int *flag)
{
	printf("ENTERED!!!!\n");
	int id = threadIdx.y;
	int closestDist = 9999999;
	int closest = -1;
	float currDist = 0;
	flag[0] = 0;
	for(int j = 0; j < num_means ; j++)
	{
		currDist = 0;
		
		for(int i = 0; i < num_attribs; i++)
		{
			currDist+= (means[j*num_attribs + i] - entries[id*num_attribs + i]) * (means[j*num_attribs + i] - entries[id*num_attribs + i]);
		}
		
		printf("Entry %d to mean %d distance : %f \n", id, j, currDist);
		
		if(currDist < closestDist)
		{
			closestDist = currDist;
			closest = j;
		}
	}
	if(closest != closestMean[id])
	{
		flag[0] = 1;
	}
	closestMean[id] = closest; 
}


int main()
{
	//Initial Declarations 
	int num_entries;
	int num_means;
	int num_attribs;

	//Read vals for init declarations
	printf("Enter the number of entries : \n");
	scanf("%d", &num_entries);
	printf("Enter the number of means : \n");
	scanf("%d", &num_means);
	printf("Enter the number of attributes : \n");
	scanf("%d", &num_attribs);

	//Utility declarations
	float means[num_means*num_attribs];
	float entries[num_entries*num_attribs];
	float distances[num_entries*num_means];
	int closestMean[num_entries];

	printf("Enter the entries : \n");
	for(int i = 0; i < num_entries*num_attribs; i++)
	{
		scanf("%f", &entries[i]);
	}

	printf("Enter the initial -- means : \n");
	for(int i = 0; i < num_means*num_attribs; i++)
	{
		scanf("%f", &means[i]);
	}

	dim3 gridCM (1,1);
	dim3 threadCM (1, num_entries);

	dim3 gridUM (1,1);
	dim3 threadUM (1, num_means);

	float *dmeans, *dentries, *ddistances;
	int *dclosestMean;
	float *dtemp;	//for UpdateMeans
	int *dflag;
	
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	hipMalloc((void **)&dmeans, 		sizeof(float)*num_means*num_attribs);
	hipMalloc((void **)&dentries, 		sizeof(float)*num_entries*num_attribs);
	hipMalloc((void **)&dclosestMean, 	sizeof(int)*num_entries);
	hipMalloc((void **)&dtemp,			sizeof(float)*num_attribs);
	hipMalloc((void **)&dflag, 			sizeof(int));
	int flag[1] = {1};
	hipMemcpy(dflag, flag, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dmeans, means, sizeof(float)*num_means*num_attribs, hipMemcpyHostToDevice);
	hipMemcpy(dentries, entries, sizeof(float)*num_entries*num_attribs, hipMemcpyHostToDevice);

	while(flag[0] == 1)
	{
		getClosestMeans<<<gridCM, threadCM>>>(dmeans, dentries, dclosestMean, num_entries, num_means, num_attribs, dflag);
		hipMemcpy(closestMean, dclosestMean, sizeof(int)*num_entries, hipMemcpyDeviceToHost);

		for(int i = 0; i < num_entries; i++)
			printf("%d -- ", closestMean[i]);
		printf("\n");

		updateMeans<<<gridUM, threadUM>>>(dmeans, dentries, dclosestMean, num_entries,num_means, num_attribs);
		hipMemcpy(flag, dflag, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(means, dmeans, sizeof(float)*num_means*num_attribs, hipMemcpyDeviceToHost);
		hipMemcpy(flag, dflag, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(dflag, flag, sizeof(int), hipMemcpyHostToDevice);
		//cudaMemcpy(dflag, flag, sizeof(int), cudaMemcpyHostToDevice);
		hipMemcpy(dmeans, means, sizeof(float)*num_means*num_attribs, hipMemcpyHostToDevice);
		hipMemcpy(dclosestMean, closestMean, sizeof(int)*num_entries, hipMemcpyDeviceToHost);
	}
		
	hipMemcpy(means, dmeans, sizeof(float)*num_means*num_attribs, hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedtime;
	hipEventElapsedTime(&elapsedtime,start,stop);


	for(int i = 0; i < num_means*num_attribs; i++)
	{
		if(i == num_attribs)
			printf("\n");

		printf("%f  ",means[i]);
	}
		printf("\nThe elapsed timer is %f\n", elapsedtime);
}
